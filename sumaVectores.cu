// Suma_Vectores.cu : 
//


#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


// Función Kernel que se ejecuta en el Device.
__global__ void Suma_vectores(float *c,float *a,float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N){
	  c[idx] = a[idx] + b[idx];
	  //printf("%d ",idx);
  }	
}

// Código principal que se ejecuta en el Host
int main(void){
	float *a_h,*b_h,*c_h; //Punteros a arreglos en el Host 
	float *a_d,*b_d,*c_d;  //Punteros a arreglos en el Device
	const int N = 24;  //Número de elementos en los arreglos  (probar 1000000)
	
	size_t size=N * sizeof(float);

	a_h = (float *)malloc(size); // Pedimos memoria en el Host
	b_h = (float *)malloc(size);
	c_h = (float *)malloc(size);//También se puede con cudaMallocHost
	
	//Inicializamos los arreglos a,b en el Host
	for (int i=0; i<N; i++){
		a_h[i] = (float)i;
		b_h[i] = (float)(i+1);
	}

	printf("\nArreglo a:\n");
	for (int i=0; i<N; i++) printf("%f ", a_h[i]);
	printf("\n\nArreglo b:\n");
	for (int i=0; i<N; i++) printf("%f ", b_h[i]);
	
	hipMalloc((void **) &a_d,size);   // Pedimos memoria en el Device
	hipMalloc((void **) &b_d,size);
	hipMalloc((void **) &c_d,size);
	
	//Pasamos los arreglos a y b del Host al Device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
	
	//Realizamos el cálculo en el Device
	int block_size =8;
	int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	
	Suma_vectores <<< n_blocks, block_size >>> (c_d,a_d,b_d,N);
		
	//Pasamos el resultado del Device al Host
	hipMemcpy(c_h, c_d, size,hipMemcpyDeviceToHost);
	
	//Resultado
	printf("\n\nArreglo c:\n");
	for (int i=0; i<N; i++) printf("%f ", c_h[i]);
	
	system("pause");
	
	// Liberamos la memoria del Host
	free(a_h); 
	free(b_h); 
	free(c_h); 

	// Liberamos la memoria del Device
	hipFree(a_d); 
	hipFree(b_d); 
	hipFree(c_d); 
	return(0);
}
